#include "hip/hip_runtime.h"

#include "gpu_core_headers.h"
#include "gpu_indexing_functions.h"

#include "GpuImage.h"
#include "template_matching_empricial_distribution.h"
#include "../constants/constants.h"

using namespace cistem::match_template as TM;

/**
 * @brief Construct a new TM_EmpiricalDistribution
 * Note: both histogram_min and histogram step must be > 0 or no histogram will be created
 * Note: the number of histogram bins is fixed by TM::histogram_number_of_points
 * 
 * @param reference_image - used to determine the size of the input images and set gpu launch configurations
 * @param histogram_min - the minimum value of the histogram
 * @param histogram_step - the step size of the histogram
 * @param n_images_to_accumulate_concurrently - the number of images to accumulate concurrently
 * 
 */
template <InputType input_type, bool per_image>
TM_EmpiricalDistribution::TM_EmpiricalDistribution(GpuImage&           reference_image,
                                                   histogram_storage_t histogram_min,
                                                   histogram_storage_t histogram_step,
                                                   int                 n_border_pixels_to_ignore_for_histogram,
                                                   int                 n_images_to_accumulate_concurrently,
                                                   hipStream_t*       calc_stream) : n_images_to_accumulate_concurrently_{n_images_to_accumulate_concurrently},
                                                                                n_border_pixels_to_ignore_for_histogram_{n_border_pixels_to_ignore_for_histogram},
                                                                                reference_image_{reference_image},
                                                                                calc_stream_{calc_stream} {

    static_assert(per_image == false, "This class does not support per image accumulation yet");

    // I suspect we'll move to bfloat16 for the input data, as it was not available at the time the
    // original code was implemented. The extended dynamic range, and ease of conversion to/from histogram_storage_t
    // are likely a benefit, while the further reduced precision is unlikely to be a problem in the raw data values.
    // If anything, given that the output of the matched filter is ~ Gaussian, all the numbers closer to zero are less
    // likely to be flushed to zero when denormal, so in that respect, bflaot16 may actually maintain higher precision.
    if constexpr ( std::is_same_v<InputType, __half> ) {
        histogram_min_  = __float2half_rn(histogram_min);
        histogram_step_ = __float2half_rn(histogram_step);
    }
    else if constexpr ( std::is_same_v<InputType, __hip_bfloat16> ) {
        histogram_min_  = __float2bfloat16_rn(histogram_min);
        histogram_step_ = __float2bfloat16_rn(histogram_step);
    }
    else if constexpr ( std::is_same_v<InputType, histogram_storage_t> ) {
        histogram_min_  = histogram_min;
        histogram_step_ = histogram_step;
    }
    else {
        MyDebugAssertTrue(false, "input_type must be either __half __hip_bfloat16, or histogram_storage_t");
    }

    if ( histogram_min_ > 0.0f && histogram_step_ > 0.0f ) {
        MyDebugAssertTrue(TM::histogram_number_of_points <= 1024, "The histogram kernel assumes <= 1024 threads per block");
        MyDebugAssertTrue(TM::histogram_number_of_points % cistem::gpu::warp_size == 0, "The histogram kernel assumes a multiple of 32 threads per block");
        histogram_n_bins_ = TM::histogram_number_of_points;
    }
    else {
        // will be used as check on which kernels to call
        histogram_n_bins_ = 0;
    }

    image_dims_.x = reference_image.dims.x;
    image_dims_.y = reference_image.dims.y;
    image_dims_.z = reference_image.dims.z;
    image_dims_.w = reference_image.dims.w;

    MyDebugAssertTrue(image_dims_.x > 0 && image_dims_.y > 0 && image_dims_.z > 0 && image_dims_.w > 0, "Image dimensions must be > 0");

    // Set-up the launch configuration - assumed to be a real space image.
    // WARNING: this is up to the developer to ensure, as we'll use pointers for the input arrays
    // Note: we prefer the "1d" grid as a NxN patch is more likely to have similar values than a N^2x1 line, and so more atomic collisions in the histogram kernel.
    reference_image_.ReturnLaunchParameters<TM::histogram_number_of_points, 1>(image_dims_, true);
    gridDims_  = reference_image_.gridDims;
    blockDims_ = reference_image_.blockDims;

    // Every block will have a shared memory array of the size of the number of bins and aggregate those into their own
    // temp arrays. Only at the end of the search will these be added together'

    // Array of temporary storage to accumulate the shared mem to
    cudaErr(hipMallocAsync(&histogram_,
                            gridDims_img.x * gridDims_img.y * TM::histogram_number_of_points * sizeof(histogram_storage_t), *calc_stream_)));
};

template <InputType input_type, bool per_image>
TM_EmpiricalDistribution::~TM_EmpiricalDistribution( ) {
    cudaErr(hipFreeAsync(histogram_, *calc_stream_));
};

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Kernels and inline helper functions called from EmpiricalDistribution::AccumulateDistribution
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
inline __device__ int convert_input(T* input_ptr, int x, int y, int NW, T bin_min, T bin_inc) {
    if constexpr ( std::is_same_v<T, __half> )
        return __half2int_rd((input_ptr[y * NW + x] - bin_min) / bin_inc);
    if constexpr ( std::is_same_v<T, __hip_bfloat16> )
        return __bfloat162int_rd((input_ptr[y * NW + x] - bin_min) / bin_inc);
    if constexpr ( std::is_same_v<T, histogram_storage_t> )
        return __float2int_rd((input_ptr[y * NW + x] - bin_min) / bin_inc);
}

template <InputType input_type>
__global__ void
histogram_smem_atomics(const InputType* __restrict__ input_ptr,
                       histogram_storage_t* output_ptr,
                       int4                 dims,
                       const InputType      bin_min,
                       const InputType      bin_inc,
                       const int            max_padding,
                       const int            n_slices_to_process) {

    // initialize temporary accumulation array input_ptr shared memory, this is equal to the number of bins input_ptr the histogram,
    // which may  be more or less than the number of threads input_ptr a block
    __shared__ int smem[TM::histogram_number_of_points];

    // Each block has it's own copy of the histogram stored input_ptr global memory, found at the linear block index
    histogram_storage_t* stored_array = &output_ptr[LinearBlockIdx_2dGrid( ) * TM::histogram_number_of_points];

    // Since the number of x-threads is enforced to be = to the number of bins, we can just copy the bins to shared memory
    // Otherwise, we would need a loop to copy the bins to shared memory e.g. -> for ( int i = threadIdx.x; i < TM::histogram_number_of_points; i += BlockDimension_2d( ) )
    smem[i] = __float2int_rn(stored_array[i]);
    __syncthreads( );

    int pixel_idx;
    int previous_pixel_idx;
    int n_counts = 0;
    // updates our block's partial histogram input_ptr shared memory

    for ( int j = max_padding + physical_Y( ); j < dims.y - max_padding; j += blockDim.y * gridDim.y ) {
        for ( int i = max_padding + physical_X( ); i < dims.x - max_padding; i += blockDim.x * gridDim.x ) {
            for ( int k = 0; k < n_slices_to_process; k++ ) {
                pixel_idx = convert_input(input_ptr, i, j, dims.w, bin_min, bin_inc);
                // we have to check n_counts first otherwise the results are undefined on the first pass.
                if ( n_counts > 0 && pixel_idx != previous_pixel_idx ) {
                    atomicAdd(&smem[previous_pixel_idx], n_counts);
                    n_counts = 0;
                }
                else {
                    n_counts++;
                }
                previous_pixel_idx = pixel_idx;
            }
        }
    }
    // We have to do a final cleanup in case we've been accumulating the same value:
    if ( n_counts > 0 ) {
        atomicAdd(&smem[previous_pixel_idx], n_counts);
    }
    __syncthreads( );

    // write partial histogram into the global memory
    // Converting to long was super slow. Given that I don't care about representing the number exactly, but do care about overflow, just switch the bins to histogram_storage_t
    // As in the read case, we would need a loop if the number of threads != number of bins e.g. -> for ( int i = threadIdx.x; i < TM::histogram_number_of_points; i += BlockDimension_2d( ) )
    stored_array[i] = __int2float_rn(smem[i]);
}

template <InputType input_type>
__global__ void
histogram_final_accum(histogram_storage_t* input_ptr, int n_bins, int n_blocks) {

    int lIDX = physical_X_1d_grid( );

    if ( lIDX < n_bins ) {
        histogram_storage_t total{0.0};
        for ( int j = 0; j < n_blocks; j++ ) {
            total += input_ptr[lIDX + n_bins * j];
        }
        // We accumulate all histograms into the first block
        input_ptr[lIDX] = total;
    }
}

/**
 * @brief Accumulate new values into the pixel wise distribution.
 * If set to record a histogram, a fused kernal will be called to accumulate the histogram and the pixel wise distribution
 * If set to track 3rd and 4th moments of the distribution, a fused kernel will be called to accumulate the moments and the pixel wise distribution
 * 
 * @param input_data - pointer to the input data to accumulate, a stack of images.
 * @param n_images_this_batch - number of slices to accumulate, must be <= n_images_to_accumulate_concurrently
 */
template <InputType input_type, bool per_image>
void TM_EmpiricalDistribution::AccumulateDistribution(InputType* input_data, int n_images_this_batch) {
    MyDebugAssertTrue(input_data, "The data to acmmulate is not input_ptr memory.");
    MyDebugAssertTrue(n_images_this_batch <= n_images_to_accumulate_concurrently_, "The number of images to accumulate is greater than the number of images to accumulate concurrently");

    if ( histogram_n_bins_ == 0 ) {
        precheck;
        histogram_smem_atomics<<<gridDims_, blockDims_, 0, *calc_stream_>>>(
                input_data,
                histogram_,
                image_dims_,
                histogram_min_,
                histogram_step_,
                n_border_pixels_to_ignore_for_histogram_);
        postcheck;
    }
    else if ( higher_order_moments_ ) {
        MyDebugAssertTrue(false, "Skew and kurtosis not implemented yet");
        // call the pixel wise kernel
    }
    else {
        MyDebugAssertFalse(true, "This should never happen");
        precheck;
        histogram_and_stats_smem_atomics<<<gridDims_, blockDims_, 0, *calc_stream_>>>(
                input_data,
                image_dims_,
                histogram_,
                histogram_min_,
                histogram_step_,
                n_border_pixels_to_ignore_for_histogram_);
        postcheck;
    }
};

template <InputType input_type, bool per_image>
void TM_EmpiricalDistribution::FinalAccumulate( ) {
    precheck;
    histogram_final_accum<<<gridDims_, blockDims_, 0, *calc_stream_>>>(histogram_, TM::histogram_number_of_points, gridDims_.x * gridDims_.y);
    postcheck;
}

template <InputType input_type, bool per_image>
void TM_EmpiricalDistribution::CopyToHostAndAdd(long* array_to_add_to) {

    // Make a temporary copy of the cummulative histogram on the host and then add on the host. TODO errorchecking
    histogram_storage_t* tmp_array;
    cudaErr(hipHostMalloc(&tmp_array, TM::histogram_number_of_points * sizeof(histogram_storage_t)));
    cudaErr(hipMemcpy(tmp_array, histogram_, TM::histogram_number_of_points * sizeof(histogram_storage_t), hipMemcpyDeviceToHost));

    for ( int iBin = 0; iBin < TM::histogram_number_of_points; iBin++ ) {
        array_to_add_to[iBin] += long(tmp_array[iBin]);
    }

    cudaErr(hipHostFree(tmp_array));
}
#include "hip/hip_runtime.h"

#include "gpu_core_headers.h"
#include "gpu_indexing_functions.h"

#include "GpuImage.h"
#include "template_matching_empirical_distribution.h"
#include "../constants/constants.h"

namespace TM = cistem::match_template;

template <typename T>
inline __device__ __host__ bool test_gt_zero(T value) {
    if constexpr ( std::is_same_v<T, __half> )
        return value > HIPRT_ZERO_FP16;
    else if constexpr ( std::is_same_v<T, __hip_bfloat16> )
        return value > CUDART_ZERO_BF16;
    else if constexpr ( std::is_same_v<T, histogram_storage_t> )
        return value > 0.0f;
    else
        MyDebugAssertTrue(false, "input_type must be either __half __hip_bfloat16, or histogram_storage_t");
}

/**
 * @brief Construct a new TM_EmpiricalDistribution
 * Note: both histogram_min and histogram step must be > 0 or no histogram will be created
 * Note: the number of histogram bins is fixed by TM::histogram_number_of_points
 * 
 * @param reference_image - used to determine the size of the input images and set gpu launch configurations
 * @param histogram_min - the minimum value of the histogram
 * @param histogram_step - the step size of the histogram
 * @param n_images_to_accumulate_concurrently - the number of images to accumulate concurrently
 * 
 */
template <typename ccfType, typename mipType, bool per_image>
TM_EmpiricalDistribution<ccfType, mipType, per_image>::TM_EmpiricalDistribution(GpuImage&           reference_image,
                                                                                histogram_storage_t histogram_min,
                                                                                histogram_storage_t histogram_step,
                                                                                int                 n_border_pixels_to_ignore_for_histogram,
                                                                                const int           n_images_to_accumulate_concurrently,
                                                                                hipStream_t        calc_stream) : n_images_to_accumulate_concurrently_{n_images_to_accumulate_concurrently},
                                                                                                            n_border_pixels_to_ignore_for_histogram_{n_border_pixels_to_ignore_for_histogram},
                                                                                                            calc_stream_{calc_stream},
                                                                                                            higher_order_moments_{false} {

    static_assert(per_image == false, "This class does not support per image accumulation yet");

    MyDebugAssertFalse(hipStreamQuery(calc_stream_) == hipErrorInvalidHandle, "The cuda stream is invalid");
    // I suspect we'll move to bfloat16 for the input data, as it was not available at the time the
    // original code was implemented. The extended dynamic range, and ease of conversion to/from histogram_storage_t
    // are likely a benefit, while the further reduced precision is unlikely to be a problem in the raw data values.
    // If anything, given that the output of the matched filter is ~ Gaussian, all the numbers closer to zero are less
    // likely to be flushed to zero when denormal, so in that respect, bflaot16 may actually maintain higher precision.
    if constexpr ( std::is_same_v<ccfType, __half> ) {
        histogram_min_  = __float2half_rn(histogram_min);
        histogram_step_ = __float2half_rn(histogram_step);
    }
    else if constexpr ( std::is_same_v<ccfType, __hip_bfloat16> ) {
        histogram_min_  = __float2bfloat16_rn(histogram_min);
        histogram_step_ = __float2bfloat16_rn(histogram_step);
    }
    else if constexpr ( std::is_same_v<ccfType, histogram_storage_t> ) {
        histogram_min_  = histogram_min;
        histogram_step_ = histogram_step;
    }
    else {
        MyDebugAssertTrue(false, "input_type must be either __half __hip_bfloat16, or histogram_storage_t");
    }

    // FIXME: this should probably be a bool rather than testing for a default zero value. Hacky habits die hard
    if ( test_gt_zero(histogram_step_) ) {
        MyDebugAssertTrue(TM::histogram_number_of_points <= 1024, "The histogram kernel assumes <= 1024 threads per block");
        MyDebugAssertTrue(TM::histogram_number_of_points % cistem::gpu::warp_size == 0, "The histogram kernel assumes a multiple of 32 threads per block");
        histogram_n_bins_ = TM::histogram_number_of_points;
    }
    else {
        // will be used as check on which kernels to call
        histogram_n_bins_ = 0;
    }

    image_dims_.x = reference_image.dims.x;
    image_dims_.y = reference_image.dims.y;
    image_dims_.z = reference_image.dims.z;
    image_dims_.w = reference_image.dims.w;

    MyDebugAssertTrue(image_dims_.x > 0 && image_dims_.y > 0 && image_dims_.z > 0 && image_dims_.w > 0, "Image dimensions must be > 0");

    // Set-up the launch configuration - assumed to be a real space image.
    // WARNING: this is up to the developer to ensure, as we'll use pointers for the input arrays
    // Note: we prefer the "1d" grid as a NxN patch is more likely to have similar values than a N^2x1 line, and so more atomic collisions in the histogram kernel.
    reference_image.ReturnLaunchParameters<TM::histogram_number_of_points, 1>(image_dims_, true);
    gridDims_        = reference_image.gridDims;
    threadsPerBlock_ = reference_image.threadsPerBlock;

    // Every block will have a shared memory array of the size of the number of bins and aggregate those into their own
    // temp arrays. Only at the end of the search will these be added together'

    // Array of temporary storage to accumulate the shared mem to
    cudaErr(hipMallocAsync(&histogram_, gridDims_.x * gridDims_.y * TM::histogram_number_of_points * sizeof(histogram_storage_t), calc_stream_));
    cudaErr(hipMemsetAsync(histogram_, 0, gridDims_.x * gridDims_.y * TM::histogram_number_of_points * sizeof(histogram_storage_t), calc_stream_));
};

template <typename ccfType, typename mipType, bool per_image>
TM_EmpiricalDistribution<ccfType, mipType, per_image>::~TM_EmpiricalDistribution( ) {
    MyDebugAssertFalse(hipStreamQuery(calc_stream_) == hipErrorInvalidHandle, "The cuda stream is invalid");

    cudaErr(hipFreeAsync(histogram_, calc_stream_));
};

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Kernels and inline helper functions called from EmpiricalDistribution::AccumulateDistribution
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
inline __device__ void convert_input(T* input_ptr, int x, int y, int z, int NY, int pitch_in_pixels, const T bin_min, const T bin_inc, int& pixel_idx, T& val, int& address) {
    address = ((z * NY + y) * pitch_in_pixels) + x;
    if constexpr ( std::is_same_v<T, __half> ) {
        val       = input_ptr[address];
        pixel_idx = __half2int_rd((val - bin_min) / bin_inc);
    }
    if constexpr ( std::is_same_v<T, __hip_bfloat16> ) {
        val       = input_ptr[address];
        pixel_idx = __bfloat162int_rd((val - bin_min) / bin_inc);
    }
    if constexpr ( std::is_same_v<T, histogram_storage_t> ) {
        val       = input_ptr[address];
        pixel_idx = __float2int_rd((val - bin_min) / bin_inc);
    }
}

template <bool evalType, typename ccfType>
inline __device__ void sum_squares_and_check_max(ccfType& val, float& sum, float& sum_sq, ccfType& max_val, int& max_idx, const int idx) {
    if constexpr ( evalType > 0 ) {
        if ( val > max_val ) {
            max_val = val;
            max_idx = idx;
        }
        if constexpr ( std::is_same_v<ccfType, __half> ) {
            sum += __half2float(val);
            sum_sq += __half2float(val) * __half2float(val);
        }
        else if constexpr ( std::is_same_v<ccfType, __hip_bfloat16> ) {
            sum += __bfloat162float(val);
            sum_sq += __bfloat162float(val) * __bfloat162float(val);
        }
        else if constexpr ( std::is_same_v<ccfType, histogram_storage_t> ) {
            sum += val;
            sum_sq += val * val;
        }
    }
}

template <bool evalType, typename ccfType, typename mipType>
inline __device__ void write_mip_and_stats(float* sum_array, float* sum_sq_array,
                                           mipType* mip_psi, mipType* theta_phi,
                                           float& sum, float& sum_sq,
                                           ccfType* psi, ccfType* theta, ccfType* phi,
                                           ccfType& max_val, int max_idx, const int address) {
    if constexpr ( evalType > 0 ) {
        sum_array[address] += sum;
        sum_sq_array[address] += sum_sq;
        sum    = 0.f;
        sum_sq = 0.f;

        if constexpr ( std::is_same_v<ccfType, __half> ) {
            if ( max_val > ccfType{-10.0} && max_val > __low2half(mip_psi[address]) ) {
                mip_psi[address]   = __halves2half2(max_val, psi[max_idx]);
                theta_phi[address] = __halves2half2(theta[max_idx], phi[max_idx]);
            }
        }
        else if constexpr ( std::is_same_v<ccfType, __hip_bfloat16> ) {
            if ( max_val > ccfType{-10.0} && max_val > __low2bfloat16(mip_psi[address]) ) {
                mip_psi[address]   = __halves2bfloat162(max_val, psi[max_idx]);
                theta_phi[address] = __halves2bfloat162(theta[max_idx], phi[max_idx]);
            }
        }
        else if constexpr ( std::is_same_v<ccfType, histogram_storage_t> ) {
            if ( max_val > ccfType{-10.0} && max_val > mip_psi[address] ) {
                mip_psi[address].x   = max_val;
                mip_psi[address].y   = psi[max_idx];
                theta_phi[address].x = theta[max_idx];
                theta_phi[address].y = phi[max_idx];
            }
        }
    }
}

// TODO: __half2 atomicAdd(__half2 *address, __half2 val);
// TODO: __hip_bfloat162 atomicAdd(__hip_bfloat162 *address, __hip_bfloat162 val);
// This would allow us to double the number of bins in the histogram, and still use atomicAdd reducing contention
template <int evalType, typename ccfType, typename mipType>
__global__ void __launch_bounds__(TM::histogram_number_of_points)
        AccumulateDistributionKernel(ccfType*             input_ptr,
                                     histogram_storage_t* output_ptr,
                                     int4                 dims,
                                     const ccfType        bin_min,
                                     const ccfType        bin_inc,
                                     const int            max_padding,
                                     const int            n_slices_to_process,
                                     float*               sum_array    = nullptr,
                                     float*               sum_sq_array = nullptr,
                                     mipType*             mip_psi      = nullptr,
                                     mipType*             mip_theta    = nullptr,
                                     ccfType*             psi          = nullptr,
                                     ccfType*             theta        = nullptr,
                                     ccfType*             phi          = nullptr) {

    // initialize temporary accumulation array input_ptr shared memory, this is equal to the number of bins input_ptr the histogram,
    // which may  be more or less than the number of threads input_ptr a block
    __shared__ int smem[TM::histogram_number_of_points];

    // Each block has it's own copy of the histogram stored input_ptr global memory, found at the linear block index
    histogram_storage_t* stored_array = &output_ptr[LinearBlockIdx_2dGrid( ) * TM::histogram_number_of_points];

    // Since the number of x-threads is enforced to be = to the number of bins, we can just copy the bins to shared memory
    // Otherwise, we would need a loop to copy the bins to shared memory e.g. ->
    //        smem[threadIdx.x] = __float2int_rn(stored_array[threadIdx.x]);
    // FIXME:     // smem[i] =
    for ( int i = threadIdx.x; i < TM::histogram_number_of_points; i += BlockDimension_2d( ) )
        smem[i] = int(stored_array[i]);

    __syncthreads( );

    int     address;
    int     pixel_idx;
    ccfType val;
    // updates our block's partial histogram input_ptr shared memory
    int     max_idx;
    ccfType max_val = ccfType{0.0};
    float   sum{0.f}, sum_sq{0.f};
    for ( int j = max_padding + physical_Y( ); j < dims.y - max_padding; j += blockDim.y * gridDim.y ) {
        for ( int i = max_padding + physical_X( ); i < dims.x - max_padding; i += blockDim.x * gridDim.x ) {
            for ( int k = 0; k < n_slices_to_process; k++ ) {
                // pixel_idx = __half2int_rd((input_ptr[j * dims.w + i] - bin_min) / bin_inc);
                convert_input(input_ptr, i, j, k, dims.y, dims.w, bin_min, bin_inc, pixel_idx, val, address);
                if ( pixel_idx >= 0 && pixel_idx < TM::histogram_number_of_points ) {
                    atomicAdd(&smem[pixel_idx], 1);
                }
                sum_squares_and_check_max<evalType>(val, sum, sum_sq, max_val, max_idx, k);
            } // loop over slices

            // Now we need to actually write out to global memory for the mip if we are doint it
            write_mip_and_stats<evalType>(sum_array, sum_sq_array, mip_psi, mip_theta, sum, sum_sq, psi, theta, phi, max_val, max_idx, address);
        }
    }

    __syncthreads( );

    // write partial histogram into the global memory
    // Converting to long was super slow. Given that I don't care about representing the number exactly, but do care about overflow, just switch the bins to histogram_storage_t
    // As in the read case, we would need a loop if the number of threads != number of bins e.g. ->
    // stored_array[threadIdx.x] = __int2float_rn(smem[threadIdx.x]);
    for ( int i = threadIdx.x; i < TM::histogram_number_of_points; i += BlockDimension_2d( ) )
        stored_array[i] = int(smem[i]);
}

__global__ void
FinalAccumulateKernel(histogram_storage_t* input_ptr, const int n_bins, const int n_blocks) {

    int lIDX = physical_X( );

    if ( lIDX < n_bins ) {
        histogram_storage_t total{0.0};
        for ( int j = 0; j < n_blocks; j++ ) {
            total += input_ptr[lIDX + n_bins * j];
        }
        // We accumulate all histograms into the first block
        input_ptr[lIDX] = total;
    }
}

/**
 * @brief Accumulate new values into the pixel wise distribution.
 * If set to record a histogram, a fused kernal will be called to accumulate the histogram and the pixel wise distribution
 * If set to track 3rd and 4th moments of the distribution, a fused kernel will be called to accumulate the moments and the pixel wise distribution
 * 
 * @param input_data - pointer to the input data to accumulate, a stack of images.
 * @param n_images_this_batch - number of slices to accumulate, must be <= n_images_to_accumulate_concurrently
 */

template <typename ccfType, typename mipType, bool per_image>
void TM_EmpiricalDistribution<ccfType, mipType, per_image>::AccumulateDistribution(ccfType* input_data, int n_images_this_batch) {
    MyDebugAssertTrue(input_data != nullptr, "The data to acmmulate is not input_ptr memory.");
    MyDebugAssertTrue(n_images_this_batch <= n_images_to_accumulate_concurrently_, "The number of images to accumulate is greater than the number of images to accumulate concurrently");
    MyDebugAssertFalse(hipStreamQuery(calc_stream_) == hipErrorInvalidHandle, "The cuda stream is invalid");

    constexpr int n_threads_in_y_or_z = 1;
    const int     y_grid_divisor      = 32; // TODO: optimize this.
    dim3          threadsPerBlock_img = dim3(TM::histogram_number_of_points, n_threads_in_y_or_z, n_threads_in_y_or_z);

    dim3 gridDims_img = dim3((image_dims_.x + threadsPerBlock_img.x - 1) / threadsPerBlock_img.x,
                             (image_dims_.y + (y_grid_divisor + threadsPerBlock_img.y) - 1) / (y_grid_divisor - 1 + threadsPerBlock_img.y), 1);

    // Instead of calculating int((value - bin_min) / bin_inc), use a fused multiply add

    if ( histogram_n_bins_ != 0 ) {
        // TODO: move eval conditions to an enum
        constexpr int only_histogram = 0;
        precheck;
        AccumulateDistributionKernel<only_histogram><<<gridDims_img, threadsPerBlock_img, 0, calc_stream_>>>(
                input_data,
                histogram_,
                image_dims_,
                histogram_min_,
                histogram_step_,
                n_border_pixels_to_ignore_for_histogram_,
                n_images_this_batch,
                sum_array,
                sum_sq_array,
                mip_psi,
                mip_theta,
                psi,
                theta,
                phi);
        postcheck;
    }
    else if ( higher_order_moments_ ) {
        MyDebugAssertTrue(false, "Skew and kurtosis not implemented yet");
        // call the pixel wise kernel
    }
    else {
        MyDebugAssertFalse(true, "The fused kernel is not yet implemented.");
        constexpr int histogram_and_mip = 1;
        precheck;

        AccumulateDistributionKernel<histogram_and_mip><<<gridDims_img, threadsPerBlock_img, 0, calc_stream_>>>(
                input_data,
                histogram_,
                image_dims_,
                histogram_min_,
                histogram_step_,
                n_border_pixels_to_ignore_for_histogram_,
                n_images_this_batch,
                sum_array,
                sum_sq_array,
                mip_psi,
                mip_theta,
                psi,
                theta,
                phi);
        postcheck;
    }
};

template <typename ccfType, typename mipType, bool per_image>
void TM_EmpiricalDistribution<ccfType, mipType, per_image>::FinalAccumulate( ) {
    MyDebugAssertFalse(hipStreamQuery(calc_stream_) == hipErrorInvalidHandle, "The cuda stream is invalid");

    const int n_blocks = gridDims_.x * gridDims_.y;
    const int n_bins   = TM::histogram_number_of_points;

    // FIXME: this is from histogram.cu, but is probably not optimal
    dim3 threadsPerBlock_accum_array = dim3(32, 1, 1);
    dim3 gridDims_accum_array        = dim3((TM::histogram_number_of_points + threadsPerBlock_accum_array.x - 1) / threadsPerBlock_accum_array.x, 1, 1);

    precheck;
    FinalAccumulateKernel<<<gridDims_accum_array, threadsPerBlock_accum_array, 0, calc_stream_>>>(histogram_, n_bins, n_blocks);
    postcheck;
}

template <typename ccfType, typename mipType, bool per_image>
void TM_EmpiricalDistribution<ccfType, mipType, per_image>::CopyToHostAndAdd(long* array_to_add_to) {

    // Make a temporary copy of the cummulative histogram on the host and then add on the host. TODO errorchecking
    histogram_storage_t* tmp_array;
    cudaErr(hipHostMalloc(&tmp_array, TM::histogram_number_of_points * sizeof(histogram_storage_t)));
    cudaErr(hipMemcpy(tmp_array, histogram_, TM::histogram_number_of_points * sizeof(histogram_storage_t), hipMemcpyDeviceToHost));

    for ( int iBin = 0; iBin < TM::histogram_number_of_points; iBin++ ) {
        array_to_add_to[iBin] += long(tmp_array[iBin]);
    }

    cudaErr(hipHostFree(tmp_array));
}

template class TM_EmpiricalDistribution<__half, __half2, false>;
template class TM_EmpiricalDistribution<__hip_bfloat16, __hip_bfloat162, false>;

// Note: we allow for float in the constructor checking, however, we don't need this for our implementation, so we won't instantiate it.
// template class TM_EmpiricalDistribution<float, per_image>;


#include <cistem_config.h>

#include "../../gpu/gpu_core_headers.h"
#include "../../gpu/GpuImage.h"

using namespace cistem;

// #include "../../include/ieee-754-half/half.hpp"

/** 
 * @brief Allow the cpu image class to use cuda functions that either allocate page locked memory or register
 * a currently (host) allocated memory region as page locked.
 * 
*/

/**
 * @brief Convenience wrapper to Image::Allocate that also allocates page locked memory. It may be more efficient to use hipHostAlloc, but since
 * we generally assume that function to be compatible with the alignment produced by fftw_malloc used in the Image class method. 
 * 
 * @param wanted_x_size 
 * @param wanted_y_size 
 * @param wanted_z_size 
 * @param is_in_real_space 
 * @param do_fft_planning 
 */

template <typename StorageBaseType>
void Image::AllocatePageLockedMemory(int wanted_x_size, int wanted_y_size, int wanted_z_size, bool is_in_real_space, bool do_fft_planning) {
    Allocate(wanted_x_size, wanted_y_size, wanted_z_size, is_in_real_space, do_fft_planning);

    if constexpr ( std::is_same_v<StorageBaseType, half_float::half> ) {
        // The current methods require the fp32 memory to be allocated prior to allocating the fp16 memory
        Allocate16fBuffer( );
        RegisterPageLockedMemory(real_values_16f);
        SetIsMemoryPageLocked(real_values_16f, true);
    }
    if constexpr ( std::is_same_v<StorageBaseType, float> ) {
        RegisterPageLockedMemory(real_values);
        SetIsMemoryPageLocked(real_values, true);
    }
}

template void Image::AllocatePageLockedMemory<float>(int wanted_x_size, int wanted_y_size, int wanted_z_size, bool is_in_real_space, bool do_fft_planning);
template void Image::AllocatePageLockedMemory<half_float::half>(int wanted_x_size, int wanted_y_size, int wanted_z_size, bool is_in_real_space, bool do_fft_planning);

template <typename StorageBaseType>
void Image::RegisterPageLockedMemory(StorageBaseType* ptr) {
    // FIXME: see if there is any real overhead in checking this compared to tracking the bools.
    // If not, remove other bools and just check.
    if ( ! is_pointer_registered(ptr) ) {
        wxMutexLocker lock(s_mutexProtectingFFTW); // the mutex will be unlocked when this object is destroyed (when it goes out of scope)
        MyDebugAssertTrue(lock.IsOk( ), "Mute locking failed");
        if constexpr ( std::is_same_v<StorageBaseType, half_float::half> ) {
            cudaErr(hipHostRegister(real_values_16f, sizeof(StorageBaseType) * real_memory_allocated, hipHostRegisterDefault));
        }
        if constexpr ( std::is_same_v<StorageBaseType, float> ) {
            cudaErr(hipHostRegister(real_values, sizeof(StorageBaseType) * real_memory_allocated, hipHostRegisterDefault));
        }
        SetIsMemoryPageLocked(ptr, true);
    }
}

template void Image::RegisterPageLockedMemory<float>(float* ptr);
template void Image::RegisterPageLockedMemory<half_float::half>(half_float::half* ptr);

template <typename StorageBaseType>
void Image::UnRegisterPageLockedMemory(StorageBaseType* ptr) {
    MyDebugAssertTrue(IsMemoryAllocated(ptr), "Image is not in memory");

    if ( IsMemoryPageLocked(ptr) ) {
        wxMutexLocker lock(s_mutexProtectingFFTW); // the mutex will be unlocked when this object is destroyed (when it goes out of scope)
        MyDebugAssertTrue(lock.IsOk( ), "Mute locking failed");
        if constexpr ( std::is_same_v<StorageBaseType, half_float::half> ) {
            cudaErr(hipHostUnregister(real_values_16f));
        }
        if constexpr ( std::is_same_v<StorageBaseType, float> ) {
            cudaErr(hipHostUnregister(real_values));
        }
        SetIsMemoryPageLocked(ptr, false);
    }
}

//Note: template <> void Image::<type> does not work, while template void Image::<type> does
template void Image::UnRegisterPageLockedMemory<float>(float* ptr);
template void Image::UnRegisterPageLockedMemory<half_float::half>(half_float::half* ptr);

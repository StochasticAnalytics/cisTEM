#include "hip/hip_runtime.h"
#include "gpu_core_headers.h"
#include "TemplateMatchingCore.h"

#ifdef ENABLE_FastFFT
#include "../ext/FastFFT/include/FastFFT.cuh"
#endif

#define CHECK_FOR_BAD_FLOATS

#define DO_HISTOGRAM true

__global__ void MipPixelWiseKernel(__half* correlation_output, __half2* my_peaks, const int numel,
                                   __half psi, __half theta, __half phi, __half2* my_stats, __half2* my_new_peaks, const bool use_fast_fft);

TemplateMatchingCore::TemplateMatchingCore( ){

};

TemplateMatchingCore::TemplateMatchingCore(int number_of_jobs) {

    Init(number_of_jobs);
};

TemplateMatchingCore::~TemplateMatchingCore( ){

        // FIXME
        //	if (is_allocated_cummulative_histogram)
        //	{
        //		cudaErr(hipFree((void *)cummulative_histogram));
        //		cudaErr(hipHostFree((void *)h_cummulative_histogram));
        //	}

};

void TemplateMatchingCore::Init(int number_of_jobs) {
    this->nThreads                        = 1;
    this->number_of_jobs_per_image_in_gui = 1;
    this->nGPUs                           = 1;
};

void TemplateMatchingCore::Init(MyApp*           parent_pointer,
                                Image&           template_reconstruction,
                                Image&           input_image,
                                Image&           current_projection,
                                float            pixel_size_search_range,
                                float            pixel_size_step,
                                float            pixel_size,
                                float            defocus_search_range,
                                float            defocus_step,
                                float            defocus1,
                                float            defocus2,
                                float            psi_max,
                                float            psi_start,
                                float            psi_step,
                                AnglesAndShifts& angles,
                                EulerSearch&     global_euler_search,
                                float            histogram_min_scaled,
                                float            histogram_step_scaled,
                                int              histogram_number_of_bins,
                                int              max_padding,
                                int              first_search_position,
                                int              last_search_position,
                                ProgressBar*     my_progress,
                                long             total_correlation_positions,
                                bool             is_running_locally,
                                bool             use_fast_fft,
                                int              number_of_global_search_images_to_save)

{

    this->first_search_position          = first_search_position;
    this->last_search_position           = last_search_position;
    this->angles                         = angles;
    this->global_euler_search            = global_euler_search;
    this->use_fast_fft                   = use_fast_fft;
    this->n_global_search_images_to_save = number_of_global_search_images_to_save;

    this->psi_start = psi_start;
    this->psi_step  = psi_step;
    this->psi_max   = psi_max;

    // It seems that I need a copy for these - 1) confirm, 2) if already copying, maybe put straight into pinned mem with cudaHostMalloc
    this->template_reconstruction.CopyFrom(&template_reconstruction);
    this->input_image.CopyFrom(&input_image);
    this->current_projection.CopyFrom(&current_projection);

    d_input_image.Init(this->input_image);
    d_input_image.CopyHostToDevice(input_image);

    d_current_projection.Init(this->current_projection);
    d_padded_reference.Allocate(d_input_image.dims.x, d_input_image.dims.y, 1, true);
    d_max_intensity_projection.Allocate(d_input_image.dims.x, d_input_image.dims.y, number_of_global_search_images_to_save, true);
    d_best_psi.Allocate(d_input_image.dims.x, d_input_image.dims.y, number_of_global_search_images_to_save, true);
    d_best_theta.Allocate(d_input_image.dims.x, d_input_image.dims.y, number_of_global_search_images_to_save, true);
    d_best_phi.Allocate(d_input_image.dims.x, d_input_image.dims.y, number_of_global_search_images_to_save, true);

    d_sum1.Allocate(d_input_image.dims.x, d_input_image.dims.y, 1, true);
    d_sumSq1.Allocate(d_input_image.dims.x, d_input_image.dims.y, 1, true);
    d_sum2.Allocate(d_input_image.dims.x, d_input_image.dims.y, 1, true);
    d_sumSq2.Allocate(d_input_image.dims.x, d_input_image.dims.y, 1, true);
    d_sum3.Allocate(d_input_image.dims.x, d_input_image.dims.y, 1, true);
    d_sumSq3.Allocate(d_input_image.dims.x, d_input_image.dims.y, 1, true);

    wxPrintf("Setting up the histogram\n\n");
    histogram.Init(histogram_number_of_bins, histogram_min_scaled, histogram_step_scaled);
    if ( max_padding > 2 ) {
        histogram.max_padding = max_padding;
    }

    this->my_progress                 = my_progress;
    this->total_correlation_positions = total_correlation_positions;
    this->is_running_locally          = is_running_locally;

    this->parent_pointer = parent_pointer;

    // For now we are only working on the inner loop, so no need to track best_defocus and best_pixel_size

    // At the outset these are all empty cpu images, so don't xfer, just allocate on gpuDev

    // Transfer the input image_memory_should_not_be_deallocated

    cudaErr(hipStreamSynchronize(hipStreamPerThread));
};

void TemplateMatchingCore::RunInnerLoop(Image& projection_filter, float c_pixel, float c_defocus, int threadIDX, long& current_correlation_position) {

    // Make sure we are starting with zeros
    d_max_intensity_projection.Zeros( );
    d_best_psi.Zeros( );
    d_best_phi.Zeros( );
    d_best_theta.Zeros( );
    d_padded_reference.Zeros( );

    d_sum1.Zeros( );
    d_sumSq1.Zeros( );
    d_sum2.Zeros( );
    d_sumSq2.Zeros( );
    d_sum3.Zeros( );
    d_sumSq3.Zeros( );

    this->c_defocus                 = c_defocus;
    this->c_pixel                   = c_pixel;
    total_number_of_cccs_calculated = 0;

    // Either do not delete the single precision, or add in a copy here so that each loop over defocus vals
    // have a copy to work with. Otherwise this will not exist on the second loop
#ifdef ENABLE_FastFFT
    // FIXME: FastFFT works on transposed 2D xforms so for testing

    if ( use_fast_fft ) {
        // FastFFT pads from the upper left corner, so we need to shift the image so the origins coinicide
        // d_input_image.SwapRealSpaceQuadrants( );
        // d_input_image.PhaseShift((-d_current_projection.physical_address_of_box_center.x),
        //                          (-d_current_projection.physical_address_of_box_center.y),
        //                          0);
        d_input_image.PhaseShift(-(d_input_image.physical_address_of_box_center.x - d_current_projection.physical_address_of_box_center.x),
                                 -(d_input_image.physical_address_of_box_center.y - d_current_projection.physical_address_of_box_center.y),
                                 0);
        // d_input_image.SwapRealSpaceQuadrants( );

        d_input_image.BackwardFFT( );

        FastFFT::FourierTransformer<float, float, float2, 2> FT;

        // TODO: overload that takes and short4's int4's instead of the individual values
        FT.SetForwardFFTPlan(input_image.logical_x_dimension, input_image.logical_y_dimension, d_input_image.logical_z_dimension, d_input_image.dims.x, d_input_image.dims.y, d_input_image.dims.z, true);
        FT.SetInverseFFTPlan(d_input_image.dims.x, d_input_image.dims.y, d_input_image.dims.z, d_input_image.dims.x, d_input_image.dims.y, d_input_image.dims.z, true);

        FT.FwdFFT(d_input_image.real_values);

        // We've done a round trip iFFT/FFT since the input image was normalized to STD 1.0, so re-normalize by 1/n
        d_input_image.is_in_real_space = false;
        d_input_image.MultiplyByConstant(sqrtf(1.f / d_input_image.number_of_real_space_pixels));
    }

#endif
    d_input_image.CopyFP32toFP16buffer(false);
    d_padded_reference.CopyFP32toFP16buffer(false);

    cudaErr(hipMalloc((void**)&my_peaks, sizeof(__half2) * d_input_image.real_memory_allocated));
    cudaErr(hipMalloc((void**)&my_new_peaks, sizeof(__half2) * d_input_image.real_memory_allocated));
    cudaErr(hipMalloc((void**)&my_stats, sizeof(__half2) * d_input_image.real_memory_allocated));
    cudaErr(hipMemset(my_peaks, 0, sizeof(__half2) * d_input_image.real_memory_allocated));
    cudaErr(hipMemset(my_new_peaks, 0, sizeof(__half2) * d_input_image.real_memory_allocated));
    if ( n_global_search_images_to_save > 1 ) {
        cudaErr(hipMalloc((void**)&secondary_peaks, sizeof(__half) * d_input_image.real_memory_allocated * n_global_search_images_to_save * 4));
        cudaErr(hipMemset(secondary_peaks, 0, sizeof(__half) * d_input_image.real_memory_allocated * n_global_search_images_to_save * 4));
    }
    //	cudaErr(hipMemset(my_stats,0,sizeof(Peaks)*d_input_image.real_memory_allocated));

    hipEvent_t projection_is_free_Event, gpu_work_is_done_Event;
    cudaErr(hipEventCreateWithFlags(&projection_is_free_Event, hipEventDisableTiming));
    cudaErr(hipEventCreateWithFlags(&gpu_work_is_done_Event, hipEventDisableTiming));

// TODO: This will probably be a member variable
#ifdef ENABLE_FastFFT

    FastFFT::FourierTransformer<float, float, float2, 2> FT;

    // TODO: overload that takes and short4's int4's instead of the individual values
    FT.SetForwardFFTPlan(current_projection.logical_x_dimension, current_projection.logical_y_dimension, current_projection.logical_z_dimension, d_padded_reference.dims.x, d_padded_reference.dims.y, d_padded_reference.dims.z, true);
    FT.SetInverseFFTPlan(d_padded_reference.dims.x, d_padded_reference.dims.y, d_padded_reference.dims.z, d_padded_reference.dims.x, d_padded_reference.dims.y, d_padded_reference.dims.z, true);

    FastFFT::KernelFunction::my_functor<float, 0, FastFFT::KernelFunction::NOOP>     noop;
    FastFFT::KernelFunction::my_functor<float, 2, FastFFT::KernelFunction::CONJ_MUL> conj_mul;

#endif

    int   ccc_counter = 0;
    int   current_search_position;
    float average_on_edge;
    float average_of_reals;
    float temp_float;
    bool  second_loop = false;
    int   thisDevice;
    hipGetDevice(&thisDevice);
    wxPrintf("Thread %d is running on device %d\n", threadIDX, thisDevice);

    //	cudaErr(hipFuncSetCacheConfig(reinterpret_cast<const void*>(SumPixelWiseKernel), hipFuncCachePreferL1));

    //	bool make_graph = true;
    //	bool first_loop_complete = false;

    for ( current_search_position = first_search_position; current_search_position <= last_search_position; current_search_position++ ) {

        if ( current_search_position % 10 == 0 ) {
            wxPrintf("Starting position %d/ %d\n", current_search_position, last_search_position);
        }

        for ( float current_psi = psi_start; current_psi <= psi_max; current_psi += psi_step ) {

            angles.Init(global_euler_search.list_of_search_parameters[current_search_position][0], global_euler_search.list_of_search_parameters[current_search_position][1], current_psi, 0.0, 0.0);
            //			current_projection.SetToConstant(0.0f); // This also sets the FFT padding to zero
            template_reconstruction.ExtractSlice(current_projection, angles, 1.0f, false);
            current_projection.complex_values[0] = 0.0f + I * 0.0f;
            current_projection.SwapRealSpaceQuadrants( );
            current_projection.MultiplyPixelWise(projection_filter);
            current_projection.BackwardFFT( );
            average_on_edge  = current_projection.ReturnAverageOfRealValuesOnEdges( );
            average_of_reals = current_projection.ReturnAverageOfRealValues( ) - average_on_edge;

            // Make sure the device has moved on to the padded projection
            cudaErr(hipStreamWaitEvent(hipStreamPerThread, projection_is_free_Event, cudaEventWaitDefault));

            // FIXME: For current TM test hack, need to leave these ops on the CPU until sorting out
            // association with non-owned GPU memory.
            //// TO THE GPU ////
            d_current_projection.CopyHostToDevice(current_projection);

            // The average in the full padded image will be different;
            average_of_reals *= ((float)d_current_projection.number_of_real_space_pixels / (float)d_padded_reference.number_of_real_space_pixels);

            d_current_projection.MultiplyByConstant(rsqrtf(d_current_projection.ReturnSumOfSquares( ) / (float)d_padded_reference.number_of_real_space_pixels - (average_of_reals * average_of_reals)));

#ifdef ENABLE_FastFFT
            if ( use_fast_fft ) {
                // FIXME:
                d_current_projection.MultiplyByConstant(1.f / (float)d_padded_reference.number_of_real_space_pixels);

                cudaErr(hipEventRecord(projection_is_free_Event, hipStreamPerThread));
                FT.Generic_Fwd_Image_Inv(d_current_projection.real_values, (float2*)d_input_image.complex_values, d_padded_reference.real_values_fp16, noop, conj_mul, noop);
                d_padded_reference.CopyFP32toFP16buffer(false);
            }
#endif
            if ( ! use_fast_fft ) {

                d_current_projection.ClipInto(&d_padded_reference, 0, false, 0, 0, 0, 0);
                cudaErr(hipEventRecord(projection_is_free_Event, hipStreamPerThread));
            }

            if ( ! use_fast_fft ) {
                // For the cpu code (MKL and FFTW) the image is multiplied by N on the forward xform, and subsequently normalized by 1/N
                // cuFFT multiplies by 1/root(N) forward and then 1/root(N) on the inverse. The input image is done on the cpu, and so has no scaling.
                // Stating false on the forward FFT leaves the ref = ref*root(N). Then we have root(N)*ref*input * root(N) (on the inverse) so we need a factor of 1/N to come out proper. This is included in BackwardFFTAfterComplexConjMul
                d_padded_reference.ForwardFFT(false);

                // FFT of template is loaded
                // FFT of target image is loaded from fp16 buffer
                // Value is converted to fp32, conjugate multiplied then scaled by 1/N
                // iFFT is then calculated by inverse cuFFT plan
                // prior to writing, this value converted fp32 back to fp16
                d_padded_reference.BackwardFFTAfterComplexConjMul(d_input_image.complex_values_fp16, true);
            }

            if ( DO_HISTOGRAM ) {
                if ( ! histogram.is_allocated_histogram ) {
                    d_padded_reference.NppInit( );
                    histogram.BufferInit(d_padded_reference.npp_ROI);
                }
                histogram.AddToHistogram(d_padded_reference);
            }

            this->MipPixelWise(__float2half_rn(current_psi), __float2half_rn(global_euler_search.list_of_search_parameters[current_search_position][1]),
                               __float2half_rn(global_euler_search.list_of_search_parameters[current_search_position][0]));

            ccc_counter++;
            total_number_of_cccs_calculated++;

            if ( ccc_counter % 10 == 0 ) {
                this->AccumulateSums(my_stats, d_sum1, d_sumSq1);
            }

            if ( ccc_counter % 100 == 0 ) {

                d_sum2.AddImage(d_sum1);
                d_sum1.Zeros( );

                d_sumSq2.AddImage(d_sumSq1);
                d_sumSq1.Zeros( );
            }

            if ( ccc_counter % 10000 == 0 ) {

                d_sum3.AddImage(d_sum2);
                d_sum2.Zeros( );

                d_sumSq3.AddImage(d_sumSq2);
                d_sumSq2.Zeros( );
            }

            current_projection.is_in_real_space = false;
            d_padded_reference.is_in_real_space = true;
            hipEventRecord(gpu_work_is_done_Event, hipStreamPerThread);

            //			first_loop_complete = true;

            if ( is_running_locally ) {
                if ( ReturnThreadNumberOfCurrentThread( ) == 0 ) {
                    current_correlation_position++;
                    if ( current_correlation_position > total_correlation_positions )
                        current_correlation_position = total_correlation_positions;
                    my_progress->Update(current_correlation_position);
                }
            }
            else {
                temp_float             = current_correlation_position;
                JobResult* temp_result = new JobResult;
                temp_result->SetResult(1, &temp_float);
                parent_pointer->AddJobToResultQueue(temp_result);
            }

        } // loop over psi angles

        // The current goal is to have only one peak per search position.
        if ( n_global_search_images_to_save > 1 )
            UpdateSecondaryPeaks( );

    } // end of outer loop euler sphere position

    wxPrintf("\t\t\ntotal number %d\n", ccc_counter);

    hipStreamWaitEvent(hipStreamPerThread, gpu_work_is_done_Event, 0);

    this->AccumulateSums(my_stats, d_sum1, d_sumSq1);

    d_sum2.AddImage(d_sum1);
    d_sumSq2.AddImage(d_sumSq1);

    d_sum3.AddImage(d_sum2);
    d_sumSq3.AddImage(d_sumSq2);

    this->MipToImage( );

    MyAssertTrue(histogram.is_allocated_histogram, "Trying to accumulate a histogram that has not been initialized!")
            histogram.Accumulate(d_padded_reference);

    cudaErr(hipStreamSynchronize(hipStreamPerThread));

    cudaErr(hipFree(my_peaks));
    cudaErr(hipFree(my_stats));
    cudaErr(hipFree(my_new_peaks));
}

void TemplateMatchingCore::MipPixelWise(__half psi, __half theta, __half phi) {

    precheck;
    // N
    d_padded_reference.ReturnLaunchParametersLimitSMs(5.f, 1024);

    MipPixelWiseKernel<<<d_padded_reference.gridDims, d_padded_reference.threadsPerBlock, 0, hipStreamPerThread>>>((__half*)d_padded_reference.real_values_16f, my_peaks,
                                                                                                                    (int)d_padded_reference.real_memory_allocated,
                                                                                                                    psi, theta, phi, my_stats, my_new_peaks, use_fast_fft);
    postcheck;
}

__global__ void MipPixelWiseKernel(__half* correlation_output, __half2* my_peaks, const int numel,
                                   __half psi, __half theta, __half phi, __half2* my_stats, __half2* my_new_peaks, const bool use_fast_fft) {

    for ( int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x ) {

#ifdef CHECK_FOR_BAD_FLOATS
        __half half_val = correlation_output[i];
        if ( half_val == HIPRT_INF_FP16 )
            half_val = half(225.0f);
        else if ( half_val == HIPRT_NAN_FP16 )
            half_val = half(226.0f);
#else
        const __half half_val = correlation_output[i];
#endif

        if ( use_fast_fft ) {
            my_stats[i] += __halves2half2(half_val, half_val * half_val);
        }
        else {

            const __half2 input  = __half2half2(half_val * __half(10000.0));
            const __half2 mulVal = __halves2half2((__half)1.0, half_val);
            //    	my_stats[i].sum = __hadd(my_stats[i].sum, half_val);
            //    	my_stats[i].sq_sum = __hfma(__half(1000.)*half_val,half_val,my_stats[i].sq_sum);
            my_stats[i] = __hfma2(input, mulVal, my_stats[i]);
        }

        // We are always reading the mip value
        // And likely often writing both that value and the updated angles.
        // If we had some minimum threshold, (5?) we could skip all these memory ops while still collecting
        // the full set of image statistics as well as histogram values.
        if ( half_val > __low2half(my_peaks[i]) ) {
            //				tmp_peak.mip = half_val;
            my_peaks[i]     = __halves2half2(half_val, psi);
            my_new_peaks[i] = __halves2half2(theta, phi);
        }
    }
    //
}

__global__ void
UpdateSecondaryPeaksKernel(__half*   secondary_peaks,
                           __half2*  my_peaks,
                           __half2*  my_new_peaks,
                           const int NY,
                           const int NX) {

    //	When returning more than one search result, the peaks are stored in a 3d array,
    // numel * n_peaks * 4 (mip, psi, theta, phi)
    int best_index = 0;
    int offset     = 0;
    for ( int img_index = blockIdx.x * blockDim.x + threadIdx.x; img_index < NX; img_index += blockDim.x * gridDim.x ) {

        best_index = NY;
        for ( int i_peak = 0; i_peak < NY; i_peak++ ) {
            // Check to see if any peak from this search position is in the top n_peaks scores
            if ( __low2half(my_peaks[img_index]) > secondary_peaks[img_index + i_peak * NX] ) {
                best_index = i_peak;
                break;
            }
        }

        // If we didn't find a better peak, this loop will not execute
        // We have a numel * n_peaks * 4 (score, psi, theta, phi) array
        for ( int worst_peak = NY - 1; worst_peak > best_index; worst_peak-- ) {
            offset = img_index + NX * worst_peak;
            // Move the worst peak down one
            secondary_peaks[offset] = secondary_peaks[offset - NX];
            // Psi
            offset += NX * NY;
            secondary_peaks[offset] = secondary_peaks[offset - NX];
            // Theta
            offset += NX * NY;
            secondary_peaks[offset] = secondary_peaks[offset - NX];
            // Phi
            offset += NX * NY;
            secondary_peaks[offset] = secondary_peaks[offset - NX];
        }
        // Now insert the new peak
        if ( best_index < NY ) {
            offset                  = img_index + best_index * NX;
            secondary_peaks[offset] = __low2half(my_peaks[img_index]);
            // Psi
            offset += NX * NY;
            secondary_peaks[offset] = __high2half(my_peaks[img_index]);
            // Theta
            offset += NX * NY;
            secondary_peaks[offset] = __low2half(my_new_peaks[img_index]);
            // Phi
            offset += NX * NY;
            secondary_peaks[offset] = __high2half(my_new_peaks[img_index]);
        }
    }
}

void TemplateMatchingCore::UpdateSecondaryPeaks( ) {

    precheck;
    // N
    d_padded_reference.ReturnLaunchParametersLimitSMs(5.f, 1024);

    UpdateSecondaryPeaksKernel<<<d_padded_reference.gridDims, d_padded_reference.threadsPerBlock, 0, hipStreamPerThread>>>((__half*)secondary_peaks,
                                                                                                                            my_peaks,
                                                                                                                            my_new_peaks,
                                                                                                                            n_global_search_images_to_save,
                                                                                                                            (int)d_padded_reference.real_memory_allocated);
    postcheck;

    // We need to reset this each outer angle search or we'll never see new maximums
    cudaErr(hipMemsetAsync(my_peaks, 0, sizeof(__half2) * d_input_image.real_memory_allocated, hipStreamPerThread));
    cudaErr(hipMemsetAsync(my_new_peaks, 0, sizeof(__half2) * d_input_image.real_memory_allocated, hipStreamPerThread));
}

__global__ void MipToImageKernel(const __half2* my_peaks,
                                 const __half2* my_new_peaks,
                                 const __half*  secondary_peaks,
                                 const int      numel,
                                 hipfftReal*     mip,
                                 hipfftReal*     psi,
                                 hipfftReal*     theta,
                                 hipfftReal*     phi,
                                 const int      n_peaks) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    if ( x >= numel )
        return;

    if ( n_peaks == 1 ) {

        mip[x]   = (hipfftReal)__low2float(my_peaks[x]);
        psi[x]   = (hipfftReal)__high2float(my_peaks[x]);
        theta[x] = (hipfftReal)__low2float(my_new_peaks[x]);
        phi[x]   = (hipfftReal)__high2float(my_new_peaks[x]);
    }
    else {
        int offset;
        for ( int iPeak = 0; iPeak < n_peaks; iPeak++ ) {
            offset = x + numel * iPeak; // out puts are NX * NY * NZ

            mip[offset]   = (hipfftReal)secondary_peaks[offset];
            psi[offset]   = (hipfftReal)secondary_peaks[offset + numel * n_peaks];
            theta[offset] = (hipfftReal)secondary_peaks[offset + numel * n_peaks * 2];
            phi[offset]   = (hipfftReal)secondary_peaks[offset + numel * n_peaks * 3];
        }
    }
}

void TemplateMatchingCore::MipToImage( ) {

    precheck;
    dim3 threadsPerBlock = dim3(1024, 1, 1);
    dim3 gridDims        = dim3((d_max_intensity_projection.real_memory_allocated + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

    MipToImageKernel<<<gridDims, threadsPerBlock, 0, hipStreamPerThread>>>(my_peaks,
                                                                            my_new_peaks,
                                                                            secondary_peaks,
                                                                            d_padded_reference.real_memory_allocated,
                                                                            d_max_intensity_projection.real_values,
                                                                            d_best_psi.real_values,
                                                                            d_best_theta.real_values,
                                                                            d_best_phi.real_values,
                                                                            n_global_search_images_to_save);
    postcheck;
}

__global__ void AccumulateSumsKernel(__half2* my_stats, const int numel, hipfftReal* sum, hipfftReal* sq_sum, const bool use_fast_fft);

void TemplateMatchingCore::AccumulateSums(__half2* my_stats, GpuImage& sum, GpuImage& sq_sum) {

    precheck;
    dim3 threadsPerBlock = dim3(1024, 1, 1);
    dim3 gridDims        = dim3((sum.real_memory_allocated + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

    AccumulateSumsKernel<<<gridDims, threadsPerBlock, 0, hipStreamPerThread>>>(my_stats, sum.real_memory_allocated, sum.real_values, sq_sum.real_values, use_fast_fft);
    postcheck;
}

__global__ void AccumulateSumsKernel(__half2* my_stats, const int numel, hipfftReal* sum, hipfftReal* sq_sum, const bool use_fast_fft) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if ( x < numel ) {
        if ( use_fast_fft ) {
            sum[x] += __low2float(my_stats[x]);
            sq_sum[x] += __high2float(my_stats[x]);
        }
        else {
            sum[x]    = __fmaf_rn(0.0001f, __low2float(my_stats[x]), sum[x]);
            sq_sum[x] = __fmaf_rn(0.0001f, __high2float(my_stats[x]), sq_sum[x]);
        }

        my_stats[x] = __halves2half2((__half)0., (__half)0.);
    }
}
